#include "hip/hip_runtime.h"
#include "../../common/book.h"

#define imin(a, b) (a < b? a : b)
const int N = 33 * 1024;
const int threadsPerBlock = 256;

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIdx = threadIdx.x;
    float temp = 0;

    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

}