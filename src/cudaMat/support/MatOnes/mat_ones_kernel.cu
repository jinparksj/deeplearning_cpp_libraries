#include "hip/hip_runtime.h"
#include "mat_ones_kernel.h"
#define BLOCK_SIZE 32

__global__ void mat_ones_kernel(const float *__restrict__ src, float *__restrict__ dst,
        int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        dst[row * n + col] = 1.0;
    }
}

void mat_ones_kernel_exec(const float *src, float *dst, int m, int n) {
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);

    mat_ones_kernel<<<grid, block>>>(src, dst, m, n);
    hipDeviceSynchronize();
}
